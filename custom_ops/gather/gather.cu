#include "hip/hip_runtime.h"
#include <paddle/extension.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// ================================ CudaAtomicAdd 源码位于"paddle/fluid/platform/device/gpu/gpu_primitives.h" ================================
#define CUDA_ATOMIC_WRAPPER(op, T) \
  __device__ __forceinline__ T CudaAtomic##op(T *address, const T val)

#define USE_CUDA_ATOMIC(op, T) \
  CUDA_ATOMIC_WRAPPER(op, T) { return atomic##op(address, val); }

// Default thread count per block(or block size).
// TODO(typhoonzero): need to benchmark against setting this value
//                    to 1024.
constexpr int PADDLE_CUDA_NUM_THREADS = 512;

// For atomicAdd.
USE_CUDA_ATOMIC(Add, float);
USE_CUDA_ATOMIC(Add, int);
USE_CUDA_ATOMIC(Add, unsigned int);
// CUDA API uses unsigned long long int, we cannot use uint64_t here.
// It because unsigned long long int is not necessarily uint64_t
USE_CUDA_ATOMIC(Add, unsigned long long int);  // NOLINT

CUDA_ATOMIC_WRAPPER(Add, int64_t) {
  // Here, we check long long int must be int64_t.
  static_assert(sizeof(int64_t) == sizeof(long long int),  // NOLINT
                "long long should be int64");
  return CudaAtomicAdd(
      reinterpret_cast<unsigned long long int *>(address),  // NOLINT
      static_cast<unsigned long long int>(val));            // NOLINT
}

#if defined(__HIPCC__) || (defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 600)
USE_CUDA_ATOMIC(Add, double);
#else
CUDA_ATOMIC_WRAPPER(Add, double) {
  unsigned long long int *address_as_ull =                  // NOLINT
      reinterpret_cast<unsigned long long int *>(address);  // NOLINT
  unsigned long long int old = *address_as_ull, assumed;    // NOLINT

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN
  } while (assumed != old);

  return __longlong_as_double(old);
}
#endif

// ================================ CudaAtomicAdd （完） ================================





#define CUDA_KERNEL_LOOP(i, n)                            \
  for (int32_t i = blockIdx.x * blockDim.x + threadIdx.x, \
               step = blockDim.x * gridDim.x;             \
       i < (n); i += step)


// 暂时不支持定义2个泛型？
// template<typename data_t, typename index_t>
template<typename data_t>
__global__ void GatherCUDAKernel(const data_t* params, const int64_t* indices,
                                 data_t* output, size_t index_size,
                                 size_t slice_size) {
  CUDA_KERNEL_LOOP(i, index_size * slice_size) {
    int indices_i = i / slice_size;
    int slice_i = i - indices_i * slice_size;  // offset inside the slice
    int64_t gather_i = indices[indices_i];
    int64_t params_i = gather_i * slice_size + slice_i;
    *(output + i) = *(params + params_i);
  }
}


template<typename data_t>
__global__ void ScatterInitCUDAKernel(const int64_t* indices, data_t* output,
                                      size_t index_size, size_t slice_size,
                                      bool overwrite) {
  CUDA_KERNEL_LOOP(i, index_size * slice_size) {
    int indices_i = i / slice_size;
    int slice_i = i - indices_i * slice_size;  // offset inside the slice
    int64_t scatter_i = indices[indices_i];
    int64_t out_i = scatter_i * slice_size + slice_i;
    *(output + out_i) = static_cast<data_t>(0);
  }
}

template<typename data_t>
__global__ void ScatterCUDAKernel(const data_t* params, const int64_t* indices,
                                  data_t* output, size_t index_size,
                                  size_t slice_size, bool overwrite) {
  CUDA_KERNEL_LOOP(i, index_size * slice_size) {
    int indices_i = i / slice_size;
    int slice_i = i - indices_i * slice_size;  // offset inside the slice
    int64_t scatter_i = indices[indices_i];
    int64_t out_i = scatter_i * slice_size + slice_i;
    if (overwrite) {
      *(output + out_i) = *(params + i);
    } else {
      CudaAtomicAdd(output + out_i, *(params + i));
    }
  }
}



std::vector<paddle::Tensor> gather_cuda_forward(const paddle::Tensor& input, const paddle::Tensor& index){
    std::vector<int64_t> input_shape = input.shape();
    std::vector<int64_t> index_shape = index.shape();
    std::vector<int64_t> output_shape;
    output_shape.push_back(index_shape[0]);
    for (int i = 1; i < input_shape.size(); i++) {
        output_shape.push_back(input_shape[i]);
    }
    int index_size = index_shape[0];

    paddle::Tensor output = paddle::Tensor(paddle::PlaceType::kGPU, output_shape);

    // slice size
    int slice_size = 1;
    for (int i = 1; i < input_shape.size(); ++i) {
        slice_size *= input_shape[i];
    }

    int block = 512;
    int n = slice_size * index_size;
    int grid = (n + block - 1) / block;

    PD_DISPATCH_FLOATING_TYPES(
        input.type(), "GatherCUDAKernel", ([&] {
            GatherCUDAKernel<data_t><<<grid, block, 0, input.stream()>>>(
                input.data<data_t>(),
                index.data<int64_t>(),
                output.mutable_data<data_t>(input.place()),
                index_size, slice_size
            );
        })
    );

    return {output};
}



template<typename data_t>
__global__ void fill_constant_kernel(data_t* x, int value,
                                         int num){
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = gid; i < num; i += blockDim.x * gridDim.x) {
        x[i] = static_cast<data_t>(value);
    }
}

std::vector<paddle::Tensor> fill_constant(paddle::Tensor& x, int value, const paddle::Tensor& shape_as_x){
    int block = 512;
    int numel = x.size();
    int grid = (numel + block - 1) / block;

    PD_DISPATCH_FLOATING_TYPES(
        shape_as_x.type(), "fill_constant_kernel", ([&] {
            fill_constant_kernel<data_t><<<grid, block, 0, shape_as_x.stream()>>>(
                x.mutable_data<data_t>(shape_as_x.place()), value,
                numel
            );
        })
    );

    return {x};
}

std::vector<paddle::Tensor> gather_cuda_backward(const paddle::Tensor& input, const paddle::Tensor& index, const paddle::Tensor& doutput){
    std::vector<int64_t> input_shape = input.shape();
    std::vector<int64_t> index_shape = index.shape();
    std::vector<int64_t> doutput_shape = doutput.shape();
    int index_size = index_shape[0];

    paddle::Tensor dinput = paddle::Tensor(paddle::PlaceType::kGPU, input_shape);
    // dinput初始化为全0。
    dinput = fill_constant(dinput, 0, input)[0];

    // slice size
    int slice_size = 1;
    for (int i = 1; i < doutput_shape.size(); ++i) {
        slice_size *= doutput_shape[i];
    }

    int block = 512;
    int n = slice_size * index_size;
    int grid = (n + block - 1) / block;

    // 为 true 时表示覆盖写；为 false 时表示累加。一定要是false。
    bool overwrite = false;

    // 累加模式，被填写的位置要初始化为全0。
    if (!overwrite) {
        PD_DISPATCH_FLOATING_TYPES(
            input.type(), "ScatterInitCUDAKernel", ([&] {
                ScatterInitCUDAKernel<data_t><<<grid, block, 0, input.stream()>>>(
                    index.data<int64_t>(),
                    dinput.mutable_data<data_t>(input.place()),
                    index_size, slice_size, overwrite
                );
            })
        );
    }

    PD_DISPATCH_FLOATING_TYPES(
        input.type(), "ScatterCUDAKernel", ([&] {
            ScatterCUDAKernel<data_t><<<grid, block, 0, input.stream()>>>(
                doutput.data<data_t>(),
                index.data<int64_t>(),
                dinput.mutable_data<data_t>(input.place()),
                index_size, slice_size, overwrite
            );
        })
    );

    return {dinput};
}



std::vector<paddle::Tensor> gather_cuda_double_backward(const paddle::Tensor& input, const paddle::Tensor& index, const paddle::Tensor& ddx){
    std::vector<int64_t> input_shape = input.shape();
    std::vector<int64_t> index_shape = index.shape();
    std::vector<int64_t> output_shape;
    output_shape.push_back(index_shape[0]);
    for (int i = 1; i < input_shape.size(); i++) {
        output_shape.push_back(input_shape[i]);
    }
    int index_size = index_shape[0];

    paddle::Tensor ddy = paddle::Tensor(paddle::PlaceType::kGPU, output_shape);

    // slice size
    int slice_size = 1;
    for (int i = 1; i < input_shape.size(); ++i) {
        slice_size *= input_shape[i];
    }

    int block = 512;
    int n = slice_size * index_size;
    int grid = (n + block - 1) / block;

    PD_DISPATCH_FLOATING_TYPES(
        input.type(), "GatherCUDAKernel", ([&] {
            GatherCUDAKernel<data_t><<<grid, block, 0, input.stream()>>>(
                ddx.data<data_t>(),
                index.data<int64_t>(),
                ddy.mutable_data<data_t>(input.place()),
                index_size, slice_size
            );
        })
    );

    return {ddy};
}


